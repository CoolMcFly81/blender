
#include <hip/hip_runtime.h>
/*
 * Copyright 2011-2013 Blender Foundation
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 * http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

/* CUDA kernel entry points */

/* device data taken from CUDA occupancy calculator */

#ifdef __CUDA_ARCH__

/* 2.0 and 2.1 */
#if __CUDA_ARCH__ == 200 || __CUDA_ARCH__ == 210
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 32768
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 8
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 63

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 32
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 40

/* 3.0 and 3.5 */
#elif __CUDA_ARCH__ == 300 || __CUDA_ARCH__ == 350
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 65536
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 16
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 63

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 63
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* 3.2 */
#elif __CUDA_ARCH__ == 320
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 32768
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 16
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 63

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 63
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* 3.7 */
#elif __CUDA_ARCH__ == 370
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 65536
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 16
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 255

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 63
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* 5.0, 5.2, 5.3, 6.0, 6.1 */
#elif __CUDA_ARCH__ >= 500
#  define CUDA_MULTIPRESSOR_MAX_REGISTERS 65536
#  define CUDA_MULTIPROCESSOR_MAX_BLOCKS 32
#  define CUDA_BLOCK_MAX_THREADS 1024
#  define CUDA_THREAD_MAX_REGISTERS 255

/* tunable parameters */
#  define CUDA_THREADS_BLOCK_WIDTH 16
#  define CUDA_KERNEL_MAX_REGISTERS 48
#  define CUDA_KERNEL_BRANCHED_MAX_REGISTERS 63

/* unknown architecture */
#else
#  error "Unknown or unsupported CUDA architecture, can't determine launch bounds"
#endif

#include "../../kernel_compat_cuda.h"
#include "../../kernel_math.h"
#include "../../kernel_types.h"
#include "../../kernel_globals.h"
#include "../../kernel_film.h"
#include "../../kernel_path.h"
#include "../../kernel_path_branched.h"
#include "../../kernel_bake.h"

#include "../../filter/filter.h"

/* compute number of threads per block and minimum blocks per multiprocessor
 * given the maximum number of registers per thread */

#define CUDA_LAUNCH_BOUNDS(threads_block_width, thread_num_registers) \
	__launch_bounds__( \
		threads_block_width*threads_block_width, \
		CUDA_MULTIPRESSOR_MAX_REGISTERS/(threads_block_width*threads_block_width*thread_num_registers) \
		)

/* sanity checks */

#if CUDA_THREADS_BLOCK_WIDTH*CUDA_THREADS_BLOCK_WIDTH > CUDA_BLOCK_MAX_THREADS
#  error "Maximum number of threads per block exceeded"
#endif

#if CUDA_MULTIPRESSOR_MAX_REGISTERS/(CUDA_THREADS_BLOCK_WIDTH*CUDA_THREADS_BLOCK_WIDTH*CUDA_KERNEL_MAX_REGISTERS) > CUDA_MULTIPROCESSOR_MAX_BLOCKS
#  error "Maximum number of blocks per multiprocessor exceeded"
#endif

#if CUDA_KERNEL_MAX_REGISTERS > CUDA_THREAD_MAX_REGISTERS
#  error "Maximum number of registers per thread exceeded"
#endif

#if CUDA_KERNEL_BRANCHED_MAX_REGISTERS > CUDA_THREAD_MAX_REGISTERS
#  error "Maximum number of registers per thread exceeded"
#endif

/* kernels */

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_path_trace(NULL, buffer, rng_state, sample, x, y, offset, stride);
}

#ifdef __BRANCHED_PATH__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_BRANCHED_MAX_REGISTERS)
kernel_cuda_branched_path_trace(float *buffer, uint *rng_state, int sample, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_branched_path_trace(NULL, buffer, rng_state, sample, x, y, offset, stride);
}
#endif

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_byte(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_byte(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_convert_to_half_float(uchar4 *rgba, float *buffer, float sample_scale, int sx, int sy, int sw, int sh, int offset, int stride)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;
	int y = sy + blockDim.y*blockIdx.y + threadIdx.y;

	if(x < sx + sw && y < sy + sh)
		kernel_film_convert_to_half_float(NULL, rgba, buffer, sample_scale, x, y, offset, stride);
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_shader(uint4 *input,
                   float4 *output,
                   float *output_luma,
                   int type,
                   int sx,
                   int sw,
                   int offset,
                   int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw) {
		kernel_shader_evaluate(NULL,
		                       input,
		                       output,
		                       output_luma,
		                       (ShaderEvalType)type, 
		                       x,
		                       sample);
	}
}

#ifdef __BAKING__
extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_bake(uint4 *input, float4 *output, int type, int filter, int sx, int sw, int offset, int sample)
{
	int x = sx + blockDim.x*blockIdx.x + threadIdx.x;

	if(x < sx + sw)
		kernel_bake_evaluate(NULL, input, output, (ShaderEvalType)type, filter, x, offset, sample);
}
#endif

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_divide_shadow(int sample, float* buffers, int4 buffer_rect, int offset, int stride, float *unfiltered, float *sampleVariance, float *sampleVarianceV, float *bufferVariance, int4 prefilter_rect)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		int tile_x[4] = {buffer_rect.x, buffer_rect.x, buffer_rect.x+buffer_rect.z, buffer_rect.x+buffer_rect.z};
		int tile_y[4] = {buffer_rect.y, buffer_rect.y, buffer_rect.y+buffer_rect.w, buffer_rect.y+buffer_rect.w};
		float *tile_buffers[9] = {NULL, NULL, NULL, NULL, buffers, NULL, NULL, NULL, NULL};
		int tile_offset[9] = {0, 0, 0, 0, offset, 0, 0, 0, 0};
		int tile_stride[9] = {0, 0, 0, 0, stride, 0, 0, 0, 0};
		kernel_filter_divide_shadow(NULL, sample, tile_buffers, x, y, tile_x, tile_y, tile_offset, tile_stride, unfiltered, sampleVariance, sampleVarianceV, bufferVariance, prefilter_rect);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_get_feature(int sample, float* buffers, int m_offset, int v_offset, int4 buffer_rect, int offset, int stride, float *mean, float *variance, int4 prefilter_rect)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		int tile_x[4] = {buffer_rect.x, buffer_rect.x, buffer_rect.x+buffer_rect.z, buffer_rect.x+buffer_rect.z};
		int tile_y[4] = {buffer_rect.y, buffer_rect.y, buffer_rect.y+buffer_rect.w, buffer_rect.y+buffer_rect.w};
		float *tile_buffers[9] = {NULL, NULL, NULL, NULL, buffers, NULL, NULL, NULL, NULL};
		int tile_offset[9] = {0, 0, 0, 0, offset, 0, 0, 0, 0};
		int tile_stride[9] = {0, 0, 0, 0, stride, 0, 0, 0, 0};
		kernel_filter_get_feature(NULL, sample, tile_buffers, m_offset, v_offset, x, y, tile_x, tile_y, tile_offset, tile_stride, mean, variance, prefilter_rect);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_non_local_means(float *noisyImage, float *weightImage, float *variance, float *filteredImage, int4 prefilter_rect, int r, int f, float a, float k_2)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_non_local_means(x, y, noisyImage, weightImage, variance, filteredImage, prefilter_rect, r, f, a, k_2);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_combine_halves(float *mean, float *variance, float *a, float *b, int4 prefilter_rect, int r)
{
	int x = prefilter_rect.x + blockDim.x*blockIdx.x + threadIdx.x;
	int y = prefilter_rect.y + blockDim.y*blockIdx.y + threadIdx.y;
	if(x < prefilter_rect.z && y < prefilter_rect.w) {
		kernel_filter_combine_halves(x, y, mean, variance, a, b, prefilter_rect, r);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_construct_transform(int sample, float const* __restrict__ buffer, float *transform, void *storage, int4 filter_area, int4 rect)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		CUDAFilterStorage *l_storage = ((CUDAFilterStorage*) storage) + y*filter_area.z + x;
		float *l_transform = transform + y*filter_area.z + x;
		kernel_filter_construct_transform(NULL, sample, buffer, x + filter_area.x, y + filter_area.y, l_transform, l_storage, rect, filter_area.z*filter_area.w, threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_estimate_bandwidths(int sample, float const* __restrict__ buffer, float const* __restrict__ transform, void *storage, int4 filter_area, int4 rect)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		CUDAFilterStorage *l_storage = ((CUDAFilterStorage*) storage) + y*filter_area.z + x;
		float const* __restrict__ l_transform = transform + y*filter_area.z + x;
		kernel_filter_estimate_bandwidths(NULL, sample, buffer, x + filter_area.x, y + filter_area.y, l_transform, l_storage, rect, filter_area.z*filter_area.w, threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_estimate_bias_variance(int sample, float const* __restrict__ buffer, float const* __restrict__ transform, void *storage, int4 filter_area, int4 rect, int candidate)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		CUDAFilterStorage *l_storage = ((CUDAFilterStorage*) storage) + y*filter_area.z + x;
		float const* __restrict__ l_transform = transform + y*filter_area.z + x;
		kernel_filter_estimate_bias_variance(NULL, sample, buffer, x + filter_area.x, y + filter_area.y, l_transform, l_storage, rect, candidate, filter_area.z*filter_area.w, threadIdx.y*blockDim.x + threadIdx.x);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_calculate_bandwidth(int sample, void *storage, int4 filter_area)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		CUDAFilterStorage *l_storage = ((CUDAFilterStorage*) storage) + y*filter_area.z + x;
		kernel_filter_calculate_bandwidth(NULL, sample, l_storage);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_final_pass_wlr(int sample, float* buffer, int offset, int stride, float const* __restrict__ transform, void *storage, float *buffers, int4 filter_area, int4 rect)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		CUDAFilterStorage *l_storage = ((CUDAFilterStorage*) storage) + y*filter_area.z + x;
		float const* __restrict__ l_transform = transform + y*filter_area.z + x;
		float weight_cache[CUDA_WEIGHT_CACHE_SIZE];
		kernel_filter_final_pass_wlr(NULL, sample, buffer, x + filter_area.x, y + filter_area.y, offset, stride, buffers, 0, make_int2(0, 0), l_storage, weight_cache, l_transform, filter_area.z*filter_area.w, filter_area, rect);
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_final_pass_nlm(int sample, float* buffer, int offset, int stride, float const* __restrict__ transform, void *storage, float *buffers, int4 filter_area, int4 rect)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		CUDAFilterStorage *l_storage = ((CUDAFilterStorage*) storage) + y*filter_area.z + x;
		float const* __restrict__ l_transform = transform + y*filter_area.z + x;
		float weight_cache[CUDA_WEIGHT_CACHE_SIZE];
		if(kernel_data.film.denoise_cross) {
			kernel_filter_final_pass_nlm(NULL, sample, buffer, x + filter_area.x, y + filter_area.y, offset, stride, buffers, 0, make_int2(0, 6), l_storage, weight_cache, l_transform, filter_area.z*filter_area.w, filter_area, rect);
			kernel_filter_final_pass_nlm(NULL, sample, buffer, x + filter_area.x, y + filter_area.y, offset, stride, buffers, 0, make_int2(6, 0), l_storage, weight_cache, l_transform, filter_area.z*filter_area.w, filter_area, rect);
		}
		else {
			kernel_filter_final_pass_nlm(NULL, sample, buffer, x + filter_area.x, y + filter_area.y, offset, stride, buffers, 0, make_int2(0, 0), l_storage, weight_cache, l_transform, filter_area.z*filter_area.w, filter_area, rect);
		}
	}
}

extern "C" __global__ void
CUDA_LAUNCH_BOUNDS(CUDA_THREADS_BLOCK_WIDTH, CUDA_KERNEL_MAX_REGISTERS)
kernel_cuda_filter_divide_combined(float *buffers, int sample, int offset, int stride, int4 filter_area)
{
	int x = blockDim.x*blockIdx.x + threadIdx.x;
	int y = blockDim.y*blockIdx.y + threadIdx.y;
	if(x < filter_area.z && y < filter_area.w) {
		kernel_filter_divide_combined(NULL, x + filter_area.x, y + filter_area.y, sample, buffers, offset, stride);
	}
}

#endif

